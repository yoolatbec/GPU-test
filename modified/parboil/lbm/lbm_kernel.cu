#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#ifndef LBM_KERNEL_CU
#define LBM_KERNEL_CU

#include "lbm.h"

/******************************************************************************/

//mark
__global__ void performStreamCollide_kernel( float* srcGrid, float* dstGrid ) 
{

	//Using some predefined macros here.  Consider this the declaration 
        //  and initialization of the variables SWEEP_X, SWEEP_Y and SWEEP_Z

        SWEEP_VAR
        SWEEP_X = threadIdx.x;
        SWEEP_Y = blockIdx.x;
        SWEEP_Z = blockIdx.y;

	//Test whether the cell is fluid or obstacle
	if( TEST_FLAG_SWEEP( srcGrid, OBSTACLE )) {
        float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
        float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
        float tempSB, tempET, tempEB, tempWT, tempWB ;

        //Load all of the input fields
        //This is a gather operation of the SCATTER preprocessor variable
            // is undefined in layout_config.h, or a "local" read otherwise
        tempC = SRC_C(srcGrid);
        tempN = SRC_N(srcGrid);
        tempS = SRC_S(srcGrid);
        tempE = SRC_E(srcGrid);
        tempW = SRC_W(srcGrid);
        tempT = SRC_T(srcGrid);
        tempB = SRC_B(srcGrid);
        tempNE= SRC_NE(srcGrid);
        tempNW= SRC_NW(srcGrid);
        tempSE = SRC_SE(srcGrid);
        tempSW = SRC_SW(srcGrid);
        tempNT = SRC_NT(srcGrid);
        tempNB = SRC_NB(srcGrid);
        tempST = SRC_ST(srcGrid);
        tempSB = SRC_SB(srcGrid);
        tempET = SRC_ET(srcGrid);
        tempEB = SRC_EB(srcGrid);
        tempWT = SRC_WT(srcGrid);
        tempWB = SRC_WB(srcGrid);

		//Swizzle the inputs: reflect any fluid coming into this cell 
		// back to where it came from
		temp_swp = tempN ; 
		tempN = tempS ; 
		tempS = temp_swp ;
		temp_swp = tempE ; 
		tempE = tempW ; 
		tempW = temp_swp;
		temp_swp = tempT ; 
		tempT = tempB ; 
		tempB = temp_swp;
		temp_swp = tempNE; 
		tempNE = tempSW ; 
		tempSW = temp_swp;
		temp_swp = tempNW; 
		tempNW = tempSE ; 
		tempSE = temp_swp;
		temp_swp = tempNT ; 
		tempNT = tempSB ; 
		tempSB = temp_swp; 
		temp_swp = tempNB ; 
		tempNB = tempST ; 
		tempST = temp_swp;
		temp_swp = tempET ; 
		tempET= tempWB ; 
		tempWB = temp_swp;
		temp_swp = tempEB ; 
		tempEB = tempWT ; 
		tempWT = temp_swp;

        DST_C ( dstGrid ) = tempC;

        DST_N ( dstGrid ) = tempN; 
        DST_S ( dstGrid ) = tempS;
        DST_E ( dstGrid ) = tempE;
        DST_W ( dstGrid ) = tempW;
        DST_T ( dstGrid ) = tempT;
        DST_B ( dstGrid ) = tempB;

        DST_NE( dstGrid ) = tempNE;
        DST_NW( dstGrid ) = tempNW;
        DST_SE( dstGrid ) = tempSE;
        DST_SW( dstGrid ) = tempSW;
        DST_NT( dstGrid ) = tempNT;
        DST_NB( dstGrid ) = tempNB;
        DST_ST( dstGrid ) = tempST;
        DST_SB( dstGrid ) = tempSB;
        DST_ET( dstGrid ) = tempET;
        DST_EB( dstGrid ) = tempEB;
        DST_WT( dstGrid ) = tempWT;
        DST_WB( dstGrid ) = tempWB;
	}
	else {
        float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
        float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
        float tempSB, tempET, tempEB, tempWT, tempWB ;

        //Load all of the input fields
        //This is a gather operation of the SCATTER preprocessor variable
            // is undefined in layout_config.h, or a "local" read otherwise
        tempC = SRC_C(srcGrid);
        tempN = SRC_N(srcGrid);
        tempS = SRC_S(srcGrid);
        tempE = SRC_E(srcGrid);
        tempW = SRC_W(srcGrid);
        tempT = SRC_T(srcGrid);
        tempB = SRC_B(srcGrid);
        tempNE= SRC_NE(srcGrid);
        tempNW= SRC_NW(srcGrid);
        tempSE = SRC_SE(srcGrid);
        tempSW = SRC_SW(srcGrid);
        tempNT = SRC_NT(srcGrid);
        tempNB = SRC_NB(srcGrid);
        tempST = SRC_ST(srcGrid);
        tempSB = SRC_SB(srcGrid);
        tempET = SRC_ET(srcGrid);
        tempEB = SRC_EB(srcGrid);
        tempWT = SRC_WT(srcGrid);
        tempWB = SRC_WB(srcGrid);
                //The math meat of LBM: ignore for optimization
	        float ux, uy, uz, rho, u2;
		float temp1, temp2, temp_base;
		rho = tempC + tempN
			+ tempS + tempE
			+ tempW + tempT
			+ tempB + tempNE
			+ tempNW + tempSE
			+ tempSW + tempNT
			+ tempNB + tempST
			+ tempSB + tempET
			+ tempEB + tempWT
			+ tempWB;

		ux = + tempE - tempW
			+ tempNE - tempNW
			+ tempSE - tempSW
			+ tempET + tempEB
			- tempWT - tempWB;
		uy = + tempN - tempS
			+ tempNE + tempNW
			- tempSE - tempSW
			+ tempNT + tempNB
			- tempST - tempSB;
		uz = + tempT - tempB
			+ tempNT - tempNB
			+ tempST - tempSB
			+ tempET - tempEB
			+ tempWT - tempWB;

		ux /= rho;
		uy /= rho;
		uz /= rho;
		if( TEST_FLAG_SWEEP( srcGrid, ACCEL )) {
			ux = 0.005f;
			uy = 0.002f;
			uz = 0.000f;
		}
		u2 = 1.5f * (ux*ux + uy*uy + uz*uz) - 1.0f;
		temp_base = OMEGA*rho;
		temp1 = DFL1*temp_base;


		//Put the output values for this cell in the shared memory
		temp_base = OMEGA*rho;
		temp1 = DFL1*temp_base;
		temp2 = 1.0f-OMEGA;
		tempC = temp2*tempC + temp1*(                                 - u2);
	        temp1 = DFL2*temp_base;	
		tempN = temp2*tempN + temp1*(       uy*(4.5f*uy       + 3.0f) - u2);
		tempS = temp2*tempS + temp1*(       uy*(4.5f*uy       - 3.0f) - u2);
		tempT = temp2*tempT + temp1*(       uz*(4.5f*uz       + 3.0f) - u2);
		tempB = temp2*tempB + temp1*(       uz*(4.5f*uz       - 3.0f) - u2);
		tempE = temp2*tempE + temp1*(       ux*(4.5f*ux       + 3.0f) - u2);
		tempW = temp2*tempW + temp1*(       ux*(4.5f*ux       - 3.0f) - u2);
		temp1 = DFL3*temp_base;
		tempNT= temp2*tempNT + temp1 *( (+uy+uz)*(4.5f*(+uy+uz) + 3.0f) - u2);
		tempNB= temp2*tempNB + temp1 *( (+uy-uz)*(4.5f*(+uy-uz) + 3.0f) - u2);
		tempST= temp2*tempST + temp1 *( (-uy+uz)*(4.5f*(-uy+uz) + 3.0f) - u2);
		tempSB= temp2*tempSB + temp1 *( (-uy-uz)*(4.5f*(-uy-uz) + 3.0f) - u2);
		tempNE = temp2*tempNE + temp1 *( (+ux+uy)*(4.5f*(+ux+uy) + 3.0f) - u2);
		tempSE = temp2*tempSE + temp1 *((+ux-uy)*(4.5f*(+ux-uy) + 3.0f) - u2);
		tempET = temp2*tempET + temp1 *( (+ux+uz)*(4.5f*(+ux+uz) + 3.0f) - u2);
		tempEB = temp2*tempEB + temp1 *( (+ux-uz)*(4.5f*(+ux-uz) + 3.0f) - u2);
		tempNW = temp2*tempNW + temp1 *( (-ux+uy)*(4.5f*(-ux+uy) + 3.0f) - u2);
		tempSW = temp2*tempSW + temp1 *( (-ux-uy)*(4.5f*(-ux-uy) + 3.0f) - u2);
		tempWT = temp2*tempWT + temp1 *( (-ux+uz)*(4.5f*(-ux+uz) + 3.0f) - u2);
		tempWB = temp2*tempWB + temp1 *( (-ux-uz)*(4.5f*(-ux-uz) + 3.0f) - u2);

        DST_C ( dstGrid ) = tempC;

        DST_N ( dstGrid ) = tempN; 
        DST_S ( dstGrid ) = tempS;
        DST_E ( dstGrid ) = tempE;
        DST_W ( dstGrid ) = tempW;
        DST_T ( dstGrid ) = tempT;
        DST_B ( dstGrid ) = tempB;

        DST_NE( dstGrid ) = tempNE;
        DST_NW( dstGrid ) = tempNW;
        DST_SE( dstGrid ) = tempSE;
        DST_SW( dstGrid ) = tempSW;
        DST_NT( dstGrid ) = tempNT;
        DST_NB( dstGrid ) = tempNB;
        DST_ST( dstGrid ) = tempST;
        DST_SB( dstGrid ) = tempSB;
        DST_ET( dstGrid ) = tempET;
        DST_EB( dstGrid ) = tempEB;
        DST_WT( dstGrid ) = tempWT;
        DST_WB( dstGrid ) = tempWB;
	}
}

//modified
//seperate
__global__ void performStreamCollide_kernel_seperate( float* srcGrid, float* dstGrid ) 
{

	//Using some predefined macros here.  Consider this the declaration 
        //  and initialization of the variables SWEEP_X, SWEEP_Y and SWEEP_Z

        SWEEP_VAR
        SWEEP_X = threadIdx.x;
        SWEEP_Y = blockIdx.x;
        SWEEP_Z = blockIdx.y;

	//Load all of the input fields
	//This is a gather operation of the SCATTER preprocessor variable
        // is undefined in layout_config.h, or a "local" read otherwise

	//Test whether the cell is fluid or obstacle
	if( TEST_FLAG_SWEEP( srcGrid, OBSTACLE )) {
		float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
		float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
		float tempSB, tempET, tempEB, tempWT, tempWB ;

		tempC = SRC_C(srcGrid);

		//Swizzle the inputs: reflect any fluid coming into this cell 
		// back to where it came from
		DST_C ( dstGrid ) = tempC;

		tempN = SRC_N(srcGrid);
		tempS = SRC_S(srcGrid);
		temp_swp = tempN ; 
		tempN = tempS ; 
		tempS = temp_swp ;
		DST_N ( dstGrid ) = tempN; 
		DST_S ( dstGrid ) = tempS;

		tempE = SRC_E(srcGrid);
		tempW = SRC_W(srcGrid);
		temp_swp = tempE ; 
		tempE = tempW ; 
		tempW = temp_swp;
		DST_E ( dstGrid ) = tempE;
		DST_W ( dstGrid ) = tempW;

		tempT = SRC_T(srcGrid);
		tempB = SRC_B(srcGrid);
		temp_swp = tempT ; 
		tempT = tempB ; 
		tempB = temp_swp;
		DST_T ( dstGrid ) = tempT;
		DST_B ( dstGrid ) = tempB;

		tempNE= SRC_NE(srcGrid);
		tempSW = SRC_SW(srcGrid);
		temp_swp = tempNE; 
		tempNE = tempSW ; 
		tempSW = temp_swp;
		DST_NE( dstGrid ) = tempNE;
		DST_SW( dstGrid ) = tempSW;
		
		tempNW= SRC_NW(srcGrid);
		tempSE = SRC_SE(srcGrid);
		temp_swp = tempNW; 
		tempNW = tempSE ; 
		tempSE = temp_swp;
		DST_SE( dstGrid ) = tempSE;
		DST_NW( dstGrid ) = tempNW;

		tempNT = SRC_NT(srcGrid);
		tempSB = SRC_SB(srcGrid);
		temp_swp = tempNT ; 
		tempNT = tempSB ; 
		tempSB = temp_swp; 
		DST_NT( dstGrid ) = tempNT;
		DST_SB( dstGrid ) = tempSB;

		tempNB = SRC_NB(srcGrid);
		tempST = SRC_ST(srcGrid);
		temp_swp = tempNB ; 
		tempNB = tempST ; 
		tempST = temp_swp;
		DST_NB( dstGrid ) = tempNB;
		DST_ST( dstGrid ) = tempST;

		tempET = SRC_ET(srcGrid);
		tempWB = SRC_WB(srcGrid);
		temp_swp = tempET ; 
		tempET= tempWB ; 
		tempWB = temp_swp;
		DST_ET( dstGrid ) = tempET;
		DST_WB( dstGrid ) = tempWB;

		tempEB = SRC_EB(srcGrid);
		tempWT = SRC_WT(srcGrid);
		temp_swp = tempEB ; 
		tempEB = tempWT ; 
		tempWT = temp_swp;
		DST_EB( dstGrid ) = tempEB;
		DST_WT( dstGrid ) = tempWT;
	}
	else {
		float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
		float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
		float tempSB, tempET, tempEB, tempWT, tempWB ;

		tempC = SRC_C(srcGrid);
		tempN = SRC_N(srcGrid);
		tempS = SRC_S(srcGrid);
		tempE = SRC_E(srcGrid);
		tempW = SRC_W(srcGrid);
		tempT = SRC_T(srcGrid);
		tempB = SRC_B(srcGrid);
		tempNE= SRC_NE(srcGrid);
		tempNW= SRC_NW(srcGrid);
		tempSE = SRC_SE(srcGrid);
		tempSW = SRC_SW(srcGrid);
		tempNT = SRC_NT(srcGrid);
		tempNB = SRC_NB(srcGrid);
		tempST = SRC_ST(srcGrid);
		tempSB = SRC_SB(srcGrid);
		tempET = SRC_ET(srcGrid);
		tempEB = SRC_EB(srcGrid);
		tempWT = SRC_WT(srcGrid);
		tempWB = SRC_WB(srcGrid);

        //The math meat of LBM: ignore for optimization
	    float ux, uy, uz, rho, u2;
		float temp1, temp2, temp_base;
		rho = tempC + tempN
			+ tempS + tempE
			+ tempW + tempT
			+ tempB + tempNE
			+ tempNW + tempSE
			+ tempSW + tempNT
			+ tempNB + tempST
			+ tempSB + tempET
			+ tempEB + tempWT
			+ tempWB;

		ux = + tempE - tempW
			+ tempNE - tempNW
			+ tempSE - tempSW
			+ tempET + tempEB
			- tempWT - tempWB;
		uy = + tempN - tempS
			+ tempNE + tempNW
			- tempSE - tempSW
			+ tempNT + tempNB
			- tempST - tempSB;
		uz = + tempT - tempB
			+ tempNT - tempNB
			+ tempST - tempSB
			+ tempET - tempEB
			+ tempWT - tempWB;

		ux /= rho;
		uy /= rho;
		uz /= rho;
		if( TEST_FLAG_SWEEP( srcGrid, ACCEL )) {
			ux = 0.005f;
			uy = 0.002f;
			uz = 0.000f;
		}
		u2 = 1.5f * (ux*ux + uy*uy + uz*uz) - 1.0f;
		temp_base = OMEGA*rho;
		temp1 = DFL1*temp_base;


		//Put the output values for this cell in the shared memory
		temp_base = OMEGA*rho;
		temp1 = DFL1*temp_base;
		temp2 = 1.0f-OMEGA;
		tempC = temp2*tempC + temp1*(                                 - u2);
		DST_C ( dstGrid ) = tempC;

	    temp1 = DFL2*temp_base;	
		tempN = temp2*tempN + temp1*(       uy*(4.5f*uy       + 3.0f) - u2);
		DST_N ( dstGrid ) = tempN;

		tempS = temp2*tempS + temp1*(       uy*(4.5f*uy       - 3.0f) - u2);
		DST_S ( dstGrid ) = tempS;

		tempT = temp2*tempT + temp1*(       uz*(4.5f*uz       + 3.0f) - u2);
		DST_T ( dstGrid ) = tempT;
		
		tempB = temp2*tempB + temp1*(       uz*(4.5f*uz       - 3.0f) - u2);
		DST_B ( dstGrid ) = tempB;
		
		tempE = temp2*tempE + temp1*(       ux*(4.5f*ux       + 3.0f) - u2);
		DST_E ( dstGrid ) = tempE;
		
		tempW = temp2*tempW + temp1*(       ux*(4.5f*ux       - 3.0f) - u2);
		DST_W ( dstGrid ) = tempW;
		
		temp1 = DFL3*temp_base;
		tempNT= temp2*tempNT + temp1 *( (+uy+uz)*(4.5f*(+uy+uz) + 3.0f) - u2);
		DST_NT( dstGrid ) = tempNT;
		
		tempNB= temp2*tempNB + temp1 *( (+uy-uz)*(4.5f*(+uy-uz) + 3.0f) - u2);
		DST_NB( dstGrid ) = tempNB;
		
		tempST= temp2*tempST + temp1 *( (-uy+uz)*(4.5f*(-uy+uz) + 3.0f) - u2);
		DST_ST( dstGrid ) = tempST;
		
		tempSB= temp2*tempSB + temp1 *( (-uy-uz)*(4.5f*(-uy-uz) + 3.0f) - u2);
		DST_SB( dstGrid ) = tempSB;
		
		tempNE = temp2*tempNE + temp1 *( (+ux+uy)*(4.5f*(+ux+uy) + 3.0f) - u2);
		DST_NE( dstGrid ) = tempNE;
		
		tempSE = temp2*tempSE + temp1 *((+ux-uy)*(4.5f*(+ux-uy) + 3.0f) - u2);
		DST_SE( dstGrid ) = tempSE;
		
		tempET = temp2*tempET + temp1 *( (+ux+uz)*(4.5f*(+ux+uz) + 3.0f) - u2);
		DST_ET( dstGrid ) = tempET;
		
		tempEB = temp2*tempEB + temp1 *( (+ux-uz)*(4.5f*(+ux-uz) + 3.0f) - u2);
		DST_EB( dstGrid ) = tempEB;
		
		tempNW = temp2*tempNW + temp1 *( (-ux+uy)*(4.5f*(-ux+uy) + 3.0f) - u2);
		DST_NW( dstGrid ) = tempNW;
		
		tempSW = temp2*tempSW + temp1 *( (-ux-uy)*(4.5f*(-ux-uy) + 3.0f) - u2);
		DST_SW( dstGrid ) = tempSW;
		
		tempWT = temp2*tempWT + temp1 *( (-ux+uz)*(4.5f*(-ux+uz) + 3.0f) - u2);
		DST_WT( dstGrid ) = tempWT;
		
		tempWB = temp2*tempWB + temp1 *( (-ux-uz)*(4.5f*(-ux-uz) + 3.0f) - u2);
		DST_WB( dstGrid ) = tempWB;
	}

	//Write the results computed above
	//This is a scatter operation of the SCATTER preprocessor variable
        // is defined in layout_config.h, or a "local" write otherwise
}

#endif // LBM_KERNEL_CU
